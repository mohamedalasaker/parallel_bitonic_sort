﻿
#include "hip/hip_runtime.h"

#include<stdlib.h>
#include <stdio.h>
#include<math.h>

__global__ void BitonicSortParallel(int *inputData,int step,int stage, int N,int choice){
    
    // calculate the index of the thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;


    if (index < N) {

        // the stride of a thread in a stage
        int stride = (int)pow(2, step - stage);
        // the sequence length in a stage 
        int seqLength = ((int)pow(2, step - stage + 1));
        
        // if the thread is active in a stage
        if (index % seqLength  < stride ) {
        
            // to cheack whether the thread should sort ascendingly or descendingly
            if (index / ((int)pow(2, step)) % 2 == choice) {
                if (inputData[index] > inputData[index + stride]) {
                    //swap
                    int tmp = inputData[index];
                    inputData[index] = inputData[index + stride];
                    inputData[index + stride] = tmp;
                }
            }
            else {
                if (inputData[index] < inputData[index + stride]) {
                    //swap
                    int tmp = inputData[index];
                    inputData[index] = inputData[index + stride];
                    inputData[index + stride] = tmp;
                }
            }
        }
    }
}


__global__ void kernal(int N, int M) {
    int i = blockIdx.y;
    int j = blockIdx.x;
    printf("i:%d j:%d\n",i, j);
}

int main(void) {
    int* a = NULL;
    hipMalloc(&a,8);
    a[0] = 5;
    a[1] = 5;
    a[2] = 5;

    printf("%d", a[0,0]);

}